
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

#define TYPE size_t  // macro for the type of the matrices
#define N 8192  // rows of the matrix
#define M 8192  // columns of the matrix
#define TILE 32  // size of the edge of a sub-block to transpose
#define INDEX(x, y, n) x*n + y  // since matrices are laid down as arrays


// computes wallclock time
double seconds() {
    struct timeval tmp;
    double sec;
    gettimeofday(&tmp, (struct timezone *)0);
    sec = tmp.tv_sec + ((double)tmp.tv_usec) / 1000000.0;
    return sec;
}

// transpose the input matrix into the output using the naive CUDA implementation, not shared memory
__global__ void naive_transpose(TYPE* in, TYPE* out) {
    int x = threadIdx.x;
    int y = blockIdx.x;
    // we map one block to each row, then we are sure this makes sense since the maximum number of
    // blocks is 65000^2, a very huge number. On the other side, the maximum number of threads per block
    // is 1024, not so big. Then, if the matrix is bigger than that, each thread will be mapped to more than 
    // one element, and this while loop makes sure the threads take care of all the elements they pertain to
    while (x < N) {
        out[INDEX(x, y, N)] = in[INDEX(y, x, N)];
        x += blockDim.x;
    }
}

// transpose the input matrix into the output using the block algorithm implemented in CUDA. Uses shared memory
// for better memory coalescing
__global__ void fast_transpose(TYPE* in, TYPE* out) {
    // 1. Allocate auxiliary buffer of size TILE x TILE
    __shared__ TYPE buffer[TILE][TILE];
    // recover unique indices
    int x = blockIdx.x * TILE + threadIdx.x;
    int y = blockIdx.y * TILE + threadIdx.y;
    // 2. Read data from input into shared memory tile
    for (int j=0; j < TILE; j+=blockDim.y) {
        buffer[threadIdx.y + j][threadIdx.x] = in[(y + j) * M + x];
    }
    // 3. Synchronize threads to make sure the whole tile is ready
    __syncthreads();
    // swap indices
    x = blockIdx.y * TILE + threadIdx.x;
    y = blockIdx.x * TILE + threadIdx.y;
    // 4. Transpose the local tile into output
    for (int j=0; j < TILE; j+=blockDim.y) {
        out[(y + j) * M + x] = buffer[threadIdx.x][threadIdx.y + j];
    }
}

int main(int argc, char* argv[]) {
    //define matrices
    TYPE* mat_cpu,* trans_cpu;
    TYPE* mat_gpu,* trans_gpu;
    double start, stop;
    // size of each matrix
    int size = N * M * sizeof(TYPE);

    // allocate on host
    mat_cpu = (TYPE*)malloc(size);
    trans_cpu = (TYPE*)malloc(size);
    // allocate on device
    hipMalloc((void**)&mat_gpu, size);
    hipMalloc((void**)&trans_gpu, size);

    // initialize matrix on host such that mat[i][j] == i + j
    for (int i=0; i < N; ++i) {
        for (int j=0; j < M; ++j) {
            mat_cpu[INDEX(i, j, N)] = (TYPE)INDEX(i, j, N);
        }
    }
    // copy result from host to device
    hipMemcpy(mat_gpu, mat_cpu, size, hipMemcpyHostToDevice);

    // launch naive transpose kernel
    int NTHREADS = atoi(argv[1]);  // the number of threads per block, input from command line
    start = seconds();
    naive_transpose<<<N, NTHREADS>>>(mat_gpu, trans_gpu);
    // since the kernel is synchronous with respect to the CPU, we need to call the
    // following function to make sure timings are sensible
    hipDeviceSynchronize();
    stop = seconds();
    double s = stop - start;  // elapsed time
    // print results
    printf("Naive transpose:\n");
    printf("Elapsed time: %f seconds\nBandwidth: %f GB/s\n", s, (double(2 * size) / 1000000000.0) / s);

    // launch fast transpose kernel
    dim3 BLOCK(TILE, NTHREADS / TILE);
    dim3 GRID(N / TILE, M / TILE);
    start = seconds();
    fast_transpose<<<GRID, BLOCK>>>(mat_gpu, trans_gpu);
    hipDeviceSynchronize();
    stop = seconds();
    s = stop - start;
    // print results
    printf("Fast transpose with shared memory:\n");
    printf("Elapsed time: %f seconds\nBandwidth: %f GB/s\n", s, (double(2 * size) / 1000000000.0) / s);

    // copy result from host to device
    hipMemcpy(trans_cpu, trans_gpu, size, hipMemcpyDeviceToHost);
    // print sample output
    /*for (int i=0; i < 10; ++i) {
        for (int j=0; j < 10; ++j) {
            printf("%zu ", trans_cpu[INDEX(i, j, M)]);
        }
        printf("\n");
    }*/

    // deallocate
    free(mat_cpu); free(trans_cpu);
    hipFree(mat_gpu); hipFree(trans_gpu);
    
    return 0;
}
